#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>
#include <cmath>

// CUDA 内核实现的 GEMV
__global__ void gemv_cuda(int M, int N, const float* A, const float* x, float* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        float sum = 0.0f;
        for (int j = 0; j < N; j++) {
            sum += A[row * N + j] * x[j];
        }
        y[row] = sum;
    }
}

// 检查 CUDA 错误
void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// 检查 cuBLAS 错误
void checkCublasError(hipblasStatus_t err) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS error: " << err << std::endl;
        exit(EXIT_FAILURE);
    }
}

// 检查两个浮点数是否近似相等
bool areEqual(float a, float b, float epsilon = 1e-5f) {
    return fabs(a - b) < epsilon;
}

int main() {
    const int M = 1000; // 行数
    const int N = 1000; // 列数

    // 初始化数据
    float* h_A = new float[M * N];
    float* h_x = new float[N];
    float* h_y_cuda = new float[M];
    float* h_y_cublas = new float[M];

    for (int i = 0; i < M * N; ++i) h_A[i] = static_cast<float>(rand() % 10) / RAND_MAX;
    for (int i = 0; i < N; ++i) h_x[i] = static_cast<float>(rand() % 10) / RAND_MAX;

    float *d_A, *d_x, *d_y_cuda, *d_y_cublas;
    checkCudaError(hipMalloc((void**)&d_A, M * N * sizeof(float)));
    checkCudaError(hipMalloc((void**)&d_x, N * sizeof(float)));
    checkCudaError(hipMalloc((void**)&d_y_cuda, M * sizeof(float)));
    checkCudaError(hipMalloc((void**)&d_y_cublas, M * sizeof(float)));

    // 拷贝数据到设备
    checkCudaError(hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

    // CUDA GEMV 热身
    gemv_cuda<<<M, 1>>>(M, N, d_A, d_x, d_y_cuda);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        // 处理错误或退出
    }    
    hipDeviceSynchronize(); // 等待内核完成

    // cuBLAS 初始化
    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle));

    // cuBLAS GEMV 热身
    float alpha = 1.0f;
    float beta = 0.0f;
    checkCublasError(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_x, 1, &beta, d_y_cublas, 1));
    hipDeviceSynchronize(); // 等待内核完成
    // 拷贝结果回主机
    checkCudaError(hipMemcpy(h_y_cuda, d_y_cuda, M * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(h_y_cublas, d_y_cublas, M * sizeof(float), hipMemcpyDeviceToHost));
    // 检查结果是否一致
    bool isEqual = true;
    for (int i = 0; i < M; ++i) {
        if (!areEqual(h_y_cuda[i], h_y_cublas[i])) {
            isEqual = false;
            std::cerr << "Mismatch at index " << i << ": CUDA result = " << h_y_cuda[i] << ", cuBLAS result = " << h_y_cublas[i] << std::endl;
            break;
        }
    }

    if (isEqual) {
        std::cout << "Results are consistent!" << std::endl;
    } else {
        std::cerr << "Results are inconsistent!" << std::endl;
    }


    // 计时变量
    double totalTimeCuda = 0.0;
    double totalTimeCublas = 0.0;
    const int iterations = 100;

    // CUDA GEMV 迭代
    for (int i = 0; i < iterations; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        gemv_cuda<<<1, M>>>(M, N, d_A, d_x, d_y_cuda);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        totalTimeCuda += std::chrono::duration<double>(end - start).count();
    }

    // cuBLAS GEMV 迭代
    for (int i = 0; i < iterations; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        checkCublasError(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_x, 1, &beta, d_y_cublas, 1));
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        totalTimeCublas += std::chrono::duration<double>(end - start).count();
    }

    // 计算平均时间
    double averageTimeCuda = totalTimeCuda / iterations;
    double averageTimeCublas = totalTimeCublas / iterations;



    // 计算带宽 (以 GB/s 为单位)
    double bandwidthCuda = (2.0 * M * N * sizeof(float)) / (averageTimeCuda * 1e9); // GB/s
    double bandwidthCublas = (2.0 * M * N * sizeof(float)) / (averageTimeCublas * 1e9); // GB/s

    std::cout << "CUDA Average time taken: " << averageTimeCuda * 1e6 << " us" << std::endl;
    std::cout << "CUDA Bandwidth: " << bandwidthCuda << " GB/s" << std::endl;

    std::cout << "cuBLAS Average time taken: " << averageTimeCublas * 1e6 << " us" << std::endl;
    std::cout << "cuBLAS Bandwidth: " << bandwidthCublas << " GB/s" << std::endl;


    // 清理资源
    checkCublasError(hipblasDestroy(handle));
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y_cuda);
    hipFree(d_y_cublas);
    delete[] h_A;
    delete[] h_x;
    delete[] h_y_cuda;
    delete[] h_y_cublas;

    return 0;
}
